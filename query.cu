// a toy program to get device property

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(){
    int dev_count;
    hipGetDeviceCount(&dev_count);
    printf("%d\n", dev_count);
    
    hipDeviceProp_t prop;
    int i;
    for (i = 0; i < dev_count; i++){
        hipGetDeviceProperties(&prop,i);
        printf("Name: %s\n", prop.name);
        printf("SM count: %d\n", prop.multiProcessorCount);
        printf("Max threads per SM: %d\n", prop.maxThreadsPerBlock);
        printf("Max threads dim x: %d\n", prop.maxThreadsDim[0]);
        printf("Max threads dim y: %d\n", prop.maxThreadsDim[1]);
        printf("Max threads dim z: %d\n", prop.maxThreadsDim[2]);
        printf("Number of warps: %d\n", prop.warpSize);
    }
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }
    return 0;
}
