// 20181010
// Yuqiong Li
// a basic CUDA function to familiarize with usage
#include<stdio.h>
#include<hip/hip_runtime.h>

// function declarations 
__global__ void vecAddKernel(float * a, float * b, float * c, unsigned int N);

// main function 
int main()
{   
    int N = 10;    // length of vector 
    float * a, * b, * c;  // a and b are vectors. c is the result
    unsigned int size = N * sizeof(float);  // number of bytes to allocate 
    a = (float *)calloc(N, sizeof(float));
    b = (float *)calloc(N, sizeof(float));

    int i = 0;
    float sum = 0;
    for (i = 0; i < N; i++){
        a[i] = (float)i / 0.23 + 1;
        b[i] = (float)i / 5.89 + 9;
        sum += a[i] + b[i];
    }

    c = (float*) malloc(size);
 
    // 1. allocate memory on CUDA
    float * d_a, * d_b, * d_c;   // device memory 
    hipError_t err1 =  hipMalloc((void **) & d_a, size);
    hipError_t err2 = hipMalloc((void **) & d_b, size);
    hipError_t err3 = hipMalloc((void **) & d_c, size);
    if (err1 != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    if (err2 != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    if (err3 != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
     
     
     
    // copy memory 
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // 2. operate on kernels 
    vecAddKernel<<<ceil(N/256.0), 256>>>(d_a, d_b, d_c, N);

    // 3. copy the results back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }
 
    float cuda_res = 0;
    for(i = 0; i < N; i++){
        printf("%.2f\t", c[i]);
        cuda_res += c[i];
    }
 
    printf("Results from host :%.2f\n", sum);
    printf("Results from device:%.2f\n", cuda_res);

    hipFree(d_a); 
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

__global__
void vecAddKernel(float * a, float * b, float * c, unsigned int N){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i<N)  c[i] = a[i] + b[i];
}
