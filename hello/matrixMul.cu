// 20181010
// Yuqiong Li
// Matrix multiplication with CUDA
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>

#define index(i, j, n) ((i) * (n) + (j))

// declare global kernel function 
__global__ void matrixMulKernel(float * a, float * b, float * c, unsigned int m, unsigned int n, unsigned int r);

int main(){
    unsigned int m = 2000, n = 2000, r = 1000;  // dimensions
    float * a, * b, * c, *temp ;  // declare matrices

    a = (float *) malloc(m * n * sizeof(float));  // a is m by n
    b = (float *) malloc(n * r * sizeof(float));  // b is n by r
    c = (float *) calloc(m * r, sizeof(float));  // c is m by r : the result matrix
    temp = (float *) calloc(m * r, sizeof(float));  // to store GPU results
    int i = 0, j = 0;
    // initializing a
    for (i = 0; i < m; i++){
        for (j = 0; j < n; j++)
            a[index(i, j, n)] = i + j;
    }
   // initializing b
    for (i = 0; i < n; i++){
        for (j = 0; j < r; j++)
            b[index(i, j, r)] = i + j + 1;
    }

    double time_taken;
    clock_t start, end;

    // CPU version
    start = clock();    
    int k = 0; 
    for (i = 0; i < m; i++){
        for (j = 0; j < r; j++){
            for (k = 0; k < n; k++)
		c[index(i, j, r)] += a[index(i, k, n)] * b[index(k, j, r)];
        }
    }
    end = clock();
    time_taken = (double) (end - start) / CLOCKS_PER_SEC;
    printf("Time taken for CPU is %.2f.\n", time_taken);

    float val = 0.0;
    for (i = 0; i < m; i++){
        for (j = 0; j < r; j++){
            val += c[index(i, j, r)];
        }
    }
    printf("Check value for CPU: sum is %.2f\n.", val);

    // 1. allocate device memory for cuda variables 
    float * d_a, * d_b, * d_c;
    hipMalloc((void **) &d_a, m * n * sizeof(float));
    hipMalloc((void **) &d_b, n * r * sizeof(float));
    hipMalloc((void **) &d_c, m * r * sizeof(float));

    // copy memory to device 
    hipMemcpy(d_a, a, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * r * sizeof(float), hipMemcpyHostToDevice);

    // 2. invoke kernel function
    dim3 blocksPerGrid(ceil(m/16.0), ceil(r/16.0), 1);
    dim3 threadsPerBlock(16, 16, 1);
    start = clock();
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, m, n, r);    
    end = clock();
    time_taken = (double) (end - start)/ CLOCKS_PER_SEC;
    printf("Time taken for GPU is %.2f\n", time_taken);

    
    // 3. copy results to device 
    hipMemcpy(temp, d_c, m * r * sizeof(float), hipMemcpyDeviceToHost);

    val = 0;
    for (i = 0; i < m; i++){
        for (j = 0; j < r; j++){
            val += temp[index(i, j, r)];
        }
    }
    
    printf("Check value for GPU: sum is %.2f\n", val);

    free(a);
    free(b);
    free(c);
    free(temp);
    hipFree(d_c);
    hipFree(d_a);
    hipFree(d_b);
    return 0;
}


__global__ void matrixMulKernel(float * a, float * b, float * c, unsigned int m, unsigned int n, unsigned int r){
    // a function to perform matrix multiplication
    // a is m by n; b is n by r; c is the result m by r
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row < m) && (col < r)){
	float pvalue = 0;
	int k = 0;
	for (k = 0; k < n; k++){
            pvalue += a[index(row, k, n)] * b[index(k, col, r)];
	}
	c[index(row, col, r)] = pvalue;
    }
}

