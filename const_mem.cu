// 20181201
// Yuqiong Li
// a basic CUDA function to test working with device constant memory
#include <stdio.h>
#include <hip/hip_runtime.h>

const unsigned int N = 10;    // size of vectors

__constant__ float const_d_a[N];  // filter in device const memory

int main()
{
    float * a, * b;  // a and b are vectors. c is the result
    a = (float *)calloc(N, sizeof(float));
    b = (float *)calloc(N, sizeof(float));

    /**************************** Exp 1: sequential ***************************/
    int i;
    int size = N * sizeof(float);
    for (i = 0; i < N; i++){
        a[i] = (float)i / 0.23 + 1;
    }


    // 1. copy a to constant memory
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(const_d_a), a, size);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipError_t err2 = hipMemcpyFromSymbol(b, HIP_SYMBOL(const_d_a), size);
    if (err2 != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    double checksum0, checksum1;
    for (i = 0; i < N; i++){
        checksum0 += a[i];
        checksum1 += b[i];
    }

    printf("Checksum for elements in host memory is %f\n.", checksum0);
    printf("Checksum for elements in constant memory is %f\n.", checksum1);

    return 0;
}
