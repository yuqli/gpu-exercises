// 20181010
// Yuqiong Li
// a basic CUDA function to familiarize with usage
#include<stdio.h>
#include<hip/hip_runtime.h>

// function declarations 
__global__ void vecAddKernel(float * a, float * b, float * c, unsigned int N);

// main function 
int main()
{   
    int N = 10;    // length of vector 
    float * a, * b, * c;  // a and b are vectors. c is the result
    unsigned int size = N * sizeof(float);  // number of bytes to allocate 
    a = (float *)calloc(N, sizeof(float));
    b = (float *)calloc(N, sizeof(float));

    int i = 0;
    float sum = 0;
    for (i = 0; i < N; i++){
        a[i] = (float)i / 0.23 + 1;
        b[i] = (float)i / 5.89 + 9;
        sum += a[i] + b[i];
    }

    c = (float*) malloc(size);
 
    // 1. allocate memory on CUDA
    float * d_a, * d_b, * d_c;   // device memory 
    hipMalloc((void **) & d_a, size);
    hipMalloc((void **) & d_b, size);
    hipMalloc((void **) & d_c, size);
     
    // copy memory 
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // 2. operate on kernels 
    vecAddKernel<<<ceil(N/256.0), 256>>>(d_a, d_b, d_c, N);

    // 3. copy the results back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }
 
    float cuda_res = 0;
    for(i = 0; i < N; i++){
        printf("%.2f\t", c[i]);
        cuda_res += c[i];
    }
 
    printf("Results from host :%.2f\n", sum);
    printf("Results from device:%.2f\n", cuda_res);

    hipFree(d_a); 
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

__global__
void vecAddKernel(float * a, float * b, float * c, unsigned int N){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i<N)  c[i] = a[i] + b[i];
}
