#include "hip/hip_runtime.h"
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include syevd_example.cpp 
 *   g++ -o a.out syevd_example.o -L/usr/local/cuda/lib64 -lcudart -lcusolver
 *
 */

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define CUDA_CHECK_ERROR( err ) (cudaCheckError( err, __FILE__, __LINE__ ))
#define CUDA_SOLVER_CHECK_ERROR( err ) (cudaSolverCheckError( err, __FILE__, __LINE__ ))

inline void cudaCheckError( hipError_t err, const char *file, int line )
{
	// CUDA error handeling from the "CUDA by example" book
	if (err != hipSuccess)
    {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

inline void cudaSolverCheckError(hipsolverStatus_t err, const char *file, int line )
{
    // cuSolver erro handling from official document examples
	if (err != HIPSOLVER_STATUS_SUCCESS)
    {
		printf( "%d in %s at line %d\n", err, file, line );
		exit( EXIT_FAILURE );
	}
}


void printMatrix(int m, int n, const double*matrix, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = matrix[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}


// Calls cuSolver to get eigen values. Wrapper around the cuda kernel
// https://docs.nvidia.com/cuda/cusolver/index.html#syevd-example1
// @in: matrix : m x m symmetrix. double[mxm], row major layout
// @out: eigen_values, double[m]
// @out: eigeh_vectors, double[m]
void eigenDecomposition(double *matrix, double *eigen_values, double *eigen_vectors, int m) {

    hipsolverHandle_t cusolverH = NULL;

    double *d_matrix = NULL;
    double *d_eigen_values = NULL;

    int *devInfo = NULL;
    double *d_work = NULL;
    int  lwork = 0;
    int info_gpu = 0;

    // step 1: create cusolver/cublas handle
    CUDA_SOLVER_CHECK_ERROR(hipsolverDnCreate(&cusolverH));

    // step 2: create device data buffers
    CUDA_CHECK_ERROR(hipMalloc ((void**)&d_matrix, sizeof(double) * m * m));
    CUDA_CHECK_ERROR(hipMalloc ((void**)&d_eigen_values, sizeof(double) * m));
    CUDA_CHECK_ERROR(hipMalloc ((void**)&devInfo, sizeof(int)));

    CUDA_CHECK_ERROR(hipMemcpy(d_matrix, matrix, sizeof(double) * m * m, hipMemcpyHostToDevice));

    // step 3: query working space of syevd
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    CUDA_SOLVER_CHECK_ERROR(hipsolverDnDsyevd_bufferSize(
        cusolverH,
        jobz,
        uplo,
        m,
        d_matrix,
        m,
        d_eigen_values,
        &lwork));

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_work, sizeof(double)*lwork));

    // step 4: compute spectrum
    CUDA_SOLVER_CHECK_ERROR(hipsolverDnDsyevd(
        cusolverH,
        jobz,
        uplo,
        m,
        d_matrix,
        m,
        d_eigen_values,
        d_work,
        lwork,
        devInfo));

    CUDA_CHECK_ERROR(hipDeviceSynchronize());

    CUDA_CHECK_ERROR(hipMemcpy(eigen_values, d_eigen_values, sizeof(double)*m, hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipMemcpy(eigen_vectors, d_matrix, sizeof(double)*m*m, hipMemcpyDeviceToHost));  // in-place computation on device
    CUDA_CHECK_ERROR(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    // check solver status
    std::cout << "after syevd: info_gpu = " << info_gpu << std::endl;
    assert(0 == info_gpu);

     
    if (d_matrix) hipFree(d_matrix);
    if (d_eigen_values) hipFree(d_eigen_values);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (cusolverH) hipsolverDnDestroy(cusolverH);

    hipDeviceReset();
} 


int main(int argc, char*argv[])
{

    /*       | 3.5 0.5 0 |
    *   matrix = | 0.5 3.5 0 |
    *       | 0   0   2 |
    *
    */
    int m = 3;
    double matrix[m*m] = { 3.5, 0.5, 0, 0.5, 3.5, 0, 0, 0, 2.0};
    double lambda[m] = { 2.0, 3.0, 4.0};

    double V[m*m]; // eigenvectors
    double W[m]; // eigenvalues

    std::cout << "matrix = (matlab base-1)\n";
    printMatrix(m, m, matrix, m, "matrix");
    printf("=====\n");

    // call eigen value function 
    eigenDecomposition(matrix, W, V, m);


    printf("eigenvalue = (matlab base-1), ascending order\n");
    for(int i = 0 ; i < m ; i++){
        printf("W[%d] = %E\n", i+1, W[i]);
    }

    printf("V = (matlab base-1)\n");
    printMatrix(m, m, V, m, "V");
    printf("=====\n");


    // step 4: check eigenvalues
    double lambda_sup = 0;
    for(int i = 0 ; i < m ; i++){
        double error = fabs( lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error)? lambda_sup : error;
    }
    printf("|lambda - W| = %E\n", lambda_sup);

    return 0;
}



